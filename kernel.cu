#include "hip/hip_runtime.h"


#include <stdio.h>

// nvcc kernel.cu -o print_capability

__global__ void addKernel(int *c, const int *a, const int *b)
{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
}

int main()
{
    hipDeviceProp_t prp;
    hipError_t cudaStatus;

    cudaStatus = hipGetDeviceProperties(&prp, /*device=*/0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return 1;
    }

    printf("Compute Capability=%d.%d\r\n", prp.major, prp.minor);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
